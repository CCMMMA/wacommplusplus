#include "hip/hip_runtime.h"
//
// Created by Ciro Giuseppe De Vita and Gennaro Mellone on 24/12/20.
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "../Config.hpp"
#include "../Particle.hpp"
#include "kernel.h"

// Returns -1 if a < 0 and 1 if a > 0
__host__ __device__ double sgn(double a) { return (a > 0) - (a < 0); }

// Computes the remainder of the division of a by p.
// https://gcc.gnu.org/onlinedocs/gfortran/MOD.html
__host__ __device__ double mod(double a, double p) { return a-p*(int)(a/p); }

// Returns the value of a with the sign of b.
// https://gcc.gnu.org/onlinedocs/gfortran/SIGN.html
__host__ __device__ double sign(double a, double b) { return abs(a)*sgn(b); }

__global__ void move(config_data *pConfigData, particle_data *pParticleData, int ocean_time_idx, int ocean_time, int s_w, int s_rho, int eta_rho, int xi_rho, double *pOceanTime, double *pMask, double *pLonRad, double *pLatRad, double *pDepthIntervals, double *pH, float *pZeta, float *pU, float *pV, float *pW, float *pAkt, int sizeSectionParticles, int numThread){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < sizeSectionParticles){
		double health0=1;

    		// Get the random flag
    		bool random=pConfigData->random;
		hiprandState state;
		if (random){
                	hiprand_init (clock64(), idx, 0, &state);
		}

    		// Get the integration time (default 30s)
    		double dti=pConfigData->dti;

    		// Get the time in seconds between two input ocean data (default 3600s, 1h)
   		double deltat=pConfigData->deltat;

    		// Ask Angelo Riccio (default 86400)
    		double tau0=pConfigData->tau0;

    		// Probability to survive (default 1.0e-4)
    		double survprob=pConfigData->survprob;

    		// Reduction Coefficient (default 1)
    		double crid=pConfigData->crid;

    		// Sedimentation velocity (m-1. default )
    		double sv=pConfigData->sv;

    		double idet=0,jdet=0,kdet=0;

    		// Number of integration intervals
    		double iint=deltat/dti;

		// For each integration interval
        	for (int t = 0; t < iint; t++) {
			// Check if the paticle is not yet active
            		if (pParticleData[idx].time > (pOceanTime[ocean_time_idx] + (t * dti))) {
                		// The particle is not already active (already emitted, but not active)
                		break;
            		}

            		// Check of the particle health is less than its probability to survive
            		if (pParticleData[idx].health < survprob) {
                		// The particle is dead
                		pParticleData[idx].health = -1;
                		// No reason to continue, exit the integration loop
                		break;
            		}

            		// Get the integer part and the fraction part of particle k
            		auto kI = (int) pParticleData[idx].k;
            		double kF = pParticleData[idx].k - kI;

            		// Get the integer part and the fraction part of particle j
            		auto jI = (int) pParticleData[idx].j;
            		double jF = pParticleData[idx].j - jI;

            		// Get the integer part and the fraction part of particle i
            		auto iI = (int) pParticleData[idx].i;
            		double iF = pParticleData[idx].i - iI;

            		// Check if the particle is out of the domain
            		if (jI < 0 || iI < 0 || jI >= eta_rho || iI >= xi_rho) {
                		// Set the particle health
                		pParticleData[idx].health = -1;

                		// no reason to continue,  exit the integration loop
                		break;
            		}

            		// Check if the particle beached
            		if (pMask[jI * xi_rho + iI] <= 0) {
                		// Set the particle health
                		pParticleData[idx].health = -1;

                		// no reason to continue,  exit the integration loop
                		break;
            		}

            		// The particle is alive!

            		// Perform the bilinear interpolation (2D) in order to get
            		// the zeta at the particle position.
            		float z1 = pZeta[ocean_time_idx * (eta_rho * xi_rho) + (jI) * xi_rho + (iI)] * (1.0 - iF) * (1.0 - jF);
	    		float z2 = pZeta[ocean_time_idx * (eta_rho * xi_rho) + (jI + 1) * xi_rho + (iI)] * (1.0 - iF) * jF;
	    		float z3 = pZeta[ocean_time_idx * (eta_rho * xi_rho) + (jI + 1) * xi_rho + (iI + 1)] * iF * jF;
	    		float z4 = pZeta[ocean_time_idx * (eta_rho * xi_rho) + (jI) * xi_rho + (iI + 1)] * iF * (1.0 - jF);

            		// The current zeta at the particle position
            		float zeta = z1 + z2 + z3 + z4;

            		// Perform the bilinear interpolation (2D) in order to get
            		// the h (depth) at the particle position.
	    		double h1 = pH[jI * xi_rho + iI] * (1.0 - iF) * (1.0 - jF);
            		double h2 = pH[(jI + 1) * xi_rho + iI] * (1.0 - iF) * jF;
            		double h3 = pH[(jI + 1) * xi_rho + (iI + 1)] * iF * jF;
            		double h4 = pH[jI * xi_rho + (iI + 1)] * iF * (1.0 - jF);

            		// The current h (depth) at the particle position
            		double h = h1 + h2 + h3 + h4;

            		// Perform the bilinear interpolation (2D) in order to get
            		// the u component of the current field in the particle position.

	    		int oy_u = (-(int)s_rho+1)*(eta_rho * xi_rho);
	    		//int oy_u = 0;
	    		float u1 = pU[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + iI) - oy_u] * (1.0 - iF) * (1.0 - jF);
            		float u2 = pU[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + iI) - oy_u] * (1.0 - iF) * jF;
            		float u3 = pU[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + (iI + 1)) - oy_u] * iF * jF;
            		float u4 = pU[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + (iI + 1)) - oy_u] * iF * (1.0 - jF);
	
            		// The current u component in the particle position
            		float uu = u1 + u2 + u3 + u4;

            		// Perform the bilinear interpolation (2D) in order to get
            		// the v component of the current field in the particle position.
	    		int oy_v = (-(int)s_rho+1)*(eta_rho * xi_rho);
	    		float v1 = pV[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + iI) - oy_v] * (1.0 - iF) * (1.0 - jF);
            		float v2 = pV[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + iI) - oy_v] * (1.0 - iF) * jF;
            		float v3 = pV[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + (iI + 1)) - oy_v] * iF * jF;
            		float v4 = pV[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + (iI + 1)) - oy_v] * iF * (1.0 - jF);

            		// The current v component in the particle position
            		float vv = v1 + v2 + v3 + v4;

            		// Perform the bilinear interpolation (3D) in order to get
            		// the w component of the current field in the particle position.
	    		int oy_w = (-(int)s_w+1)*(eta_rho * xi_rho);
	    		float w1 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + iI) - oy_w] * (1.0 - iF) * (1.0 - jF) * (1.0 - kF);
            		float w2 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + iI) - oy_w] * (1.0 - iF) * jF * (1.0 - kF);
            		float w3 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + (iI + 1)) - oy_w] * iF * jF * (1.0 - kF);
            		float w4 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + (iI + 1)) - oy_w] * iF * (1.0 - jF) * (1.0 - kF);
	    		float w5 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + jI * xi_rho + iI) - oy_w] * (1.0 - iF) * (1.0 - jF) * kF;
            		float w6 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + iI) - oy_w] * (1.0 - iF) * jF * kF;
            		float w7 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + (iI + 1)) - oy_w] * iF * jF * kF;
            		float w8 = pW[(ocean_time_idx * (s_w * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + jI * xi_rho + (iI + 1)) - oy_w] * iF * (1.0 - jF) * kF;

            		// The current w component in the particle position
            		float ww = w1 + w2 + w3 + w4 + w5 + w6 + w7 + w8;

            		// Perform the bilinear interpolation (3D) in order to get
            		// the akt in the particle position.
	    		int oy_a = (-(int)s_w+1)*(eta_rho * xi_rho);
	    		float a1 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + iI) - oy_a] * (1.0 - iF) * (1.0 - jF) * (1.0 - kF);
            		float a2 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + iI) - oy_a] * (1.0 - iF) * jF * (1.0 - kF);
            		float a3 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + (iI + 1)) - oy_a] * iF * jF * (1.0 - kF);
            		float a4 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + kI * (eta_rho *  xi_rho) + jI * xi_rho + (iI + 1)) - oy_a] * iF * (1.0 - jF) * (1.0 - kF);
            		float a5 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + jI * xi_rho + iI) - oy_a] * (1.0 - iF) * (1.0 - jF) * kF;
            		float a6 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + iI) - oy_a] * (1.0 - iF) * jF * kF;
            		float a7 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + (jI + 1) * xi_rho + (iI + 1)) - oy_a] * iF * jF * kF;
            		float a8 = pAkt[(ocean_time_idx * (s_rho * eta_rho * xi_rho) + (kI - 1) * (eta_rho *  xi_rho) + jI * xi_rho + (iI + 1)) - oy_a] * iF * (1.0 - jF) * kF;

            		// The AKT at the particle position.
            		float aa = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;

            		// Evaluate the particle leap due to the current field (deterministic leap).
            		double dileap = uu * dti;
            		double djleap = vv * dti;
            		double dkleap = (sv + ww) * dti;

            		// Calculation of sigma profile
            		// sigmaPROF=sigma(Ixx,Iyy)*(1-Zdet/(-H(Ixx,Iyy))) ! Here is H
            		//double sigmaProf=oceanModelAdapter->sigma()(jI, iI)*(1-kdet/(-oceanModelAdapter->H()(jI,iI)))
            		double sigmaprof = 3.46 * (1 + pParticleData[idx].k / s_w);

            		// Extract 3 pseudorandom numbers
            		double gi = 0, gj = 0, gk = 0;
			if (random) {
            			for (int a = 0; a < 12; a++) {
                			gi = gi + hiprand_uniform( &state ) - 0.5;
                			gj = gj + hiprand_uniform( &state ) - 0.5;
                			gk = gk + hiprand_uniform( &state ) - 0.5;
            			}
        		}
			

            		// Random leap
            		double rileap = gi * sigmaprof;
            		double rjleap = gj * sigmaprof;
            		double rkleap = gk * aa * crid;

            		// Final leap
            		double ileap = dileap + rileap;
            		double jleap = djleap + rjleap;
            		double kleap = dkleap + rkleap;

            		double d1,d2,dd,jidist, kdist;

            		// Calculate the distance in radiants of latitude between the grid cell where is
            		// currently located the particle and the next one.
            		d1=pLatRad[xi_rho*(jI+1)+iI]-pLatRad[xi_rho*jI+iI];

            		// Calculate the distance in radiants of longitude between the grid cell where is
            		// currently located the particle and the next one.
            		d2=pLonRad[xi_rho*jI+iI+1]-pLonRad[xi_rho*jI+iI];

            		// Calculate the grid cell diagonal horizontal size using the Haversine method
            		// https://www.movable-type.co.uk/scripts/latlong.html
            		dd=pow(sin(0.5*d1),2) +
               		   pow(sin(0.5*d2),2)*
               		   cos(pLatRad[xi_rho*(jI+1)+iI])*
               		   cos(pLatRad[xi_rho*(jI  )+iI]);
            		jidist=2.0*atan2(pow(dd,.5),pow(1.0-dd,.5))*6371.0;

            		kdist=pDepthIntervals[kI-(-(int)s_w+2)]*(h+zeta);
            		if ( abs(kleap) > abs(kdist) ) {
                		kleap=sign(kdist,kleap);
            		}

            		// Calculate the new particle j candidate
            		jdet=pParticleData[idx].j+0.001*jleap/jidist;

            		// Calculate the new particle i candidate
            		idet=pParticleData[idx].i+0.001*ileap/jidist;

            		// Calculate the new particle k candidate
            		kdet=pParticleData[idx].k+kleap/kdist;

            		// Reflect if out-of-column
            		// Check if the new k have to be limited by the sealfoor
            		if ( kdet < (-(int)s_w+2)) {
                		// Limit it on the bottom
                		kdet=2.0*(-(int)s_w+2)-kdet;
            		}

            		// Check if the new k have to be limited by the seafloor
            		if ( kdet > 0. ) {
                		// Limit it on the surface
                		kdet=-kdet;
            		}

            		// Reflect if crossed the coastline

            		// Calculate the integer part of the j and i candidates
            		int jdetI=(int)(jdet);
            		int idetI=(int)(idet);

            		// Check if the candidate position is within the domain
            		if (jdetI>= 0 && idetI >= 0 && jdetI<eta_rho && idetI <xi_rho) {
                		// Check if the candidate new particle position is on land (mask=0)
                		if (pMask[xi_rho*jdetI+idetI] <= 0.0) {
                    			// Reflect the particle
                    			if (idetI < iI) {
                        			idet = (double) iI + abs(pParticleData[idx].i - idet);
                    			} else if (idetI > iI) {
                        			idet = (double) idetI - mod(idet, 1.0);
                    			}
                    			if (jdetI < jdet) {
                       	 			jdet = (double) jdetI + abs(pParticleData[idx].j - jdet);
                    			} else if (jdetI > jI) {
                        			jdet = (double) jdetI - mod(jdet, 1.0);
                    			}
                		}
            		}


            		// Assign the new particle position
            		pParticleData[idx].i=idet;
            		pParticleData[idx].j=jdet;
            		pParticleData[idx].k=kdet;

            		// Update the paticle age
            		pParticleData[idx].age=pParticleData[idx].age+dti;
            		// Decay the particle
            		pParticleData[idx].health=health0*exp(-pParticleData[idx].age/tau0);
        	}
	}
}

hipError_t cudaMoveParticle(config_data *pConfigData, particle_data *pParticleData, int ocean_time_idx, int ocean_time, int s_w, int s_rho, int eta_rho, int xi_rho, double *pOceanTime, double *pMask, double *pLonRad, double *pLatRad, double *pDepthIntervals, double *pH, float *pZeta, float *pU, float *pV, float *pW, float *pAkt, int sizeSectionParticles, int numThread, int numGPU){

    dim3 nBlocks, nThreadPerBlock = 512;

    nBlocks = sizeSectionParticles/nThreadPerBlock.x + ((sizeSectionParticles%nThreadPerBlock.x) == 0?0:1);

    move<<<nBlocks, nThreadPerBlock>>>(pConfigData, pParticleData, ocean_time_idx, ocean_time, s_w, s_rho, eta_rho, xi_rho, pOceanTime, pMask, pLonRad, pLatRad, pDepthIntervals, pH, pZeta, pU, pV, pW, pAkt, sizeSectionParticles, numThread);

    hipDeviceSynchronize();

	return hipGetLastError();
}
